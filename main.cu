#include "hip/hip_runtime.h"
#include <vector>
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <>


#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}



/***** Host functions *****/
inline bool compare(int a, int b, bool descending = true)
{
    return (a < b&& descending) || (a > b && !descending);
}


void mergeSort(int* a, int n, bool descending = true)
{
    int stride = n >> 1;
    int t = 0;
    for (int i = 0, j = stride; i < stride; i++, j++)
    {
        if (compare(a[i], a[j], descending))
        {
            t = a[j];
            a[j] = a[i];
            a[i] = t;
        }
    }
    if (stride >= 2)
    {
        mergeSort(a, stride, descending);
        mergeSort(a + stride, stride, descending);
    }
}


void hBitonicSortRecursive(int* a, int n, bool descending)
{
    int stride = 2;
    int inter_step = 1;
    while (stride <= n)
    {
        inter_step = (stride << 1);
        // Order
        for (int i = 0; i < n; i += inter_step)
        {
            mergeSort(a + i, stride, descending);
        }
        // Reverse order
        for (int i = stride; i < n; i += inter_step)
        {
            mergeSort(a + i, stride, !descending);
        }
        stride = inter_step;
    }
}


void hBitonicSort(int* a, int n, bool descending)
{
    int t = 0;
    int half_stride = 1, hs = 1, s = 2;
    int hn = n >> 1;
    for (int stride = 2; stride <= n; stride <<= 1)
    {
        s = stride;
        while (s >= 2)
        {
            hs = s >> 1;
            for (int i = 0; i < hn; i++)
            {
                bool orange = (i / half_stride) % 2 == 0;
                int j = (i / hs) * s + (i % hs);
                int k = j + hs;
                //printf("Stride: %d, s: %d, i: %d, j: %d, k: %d\n", stride, s, i, j, k);
                if ((descending && ((orange && a[j] < a[k]) || (!orange && a[j] > a[k]))) ||
                    (!descending && ((orange && a[j] > a[k]) || (!orange && a[j] < a[k]))))
                {
                    t = a[k];
                    a[k] = a[j];
                    a[j] = t;
                }
            }
            s = hs;
        }
        half_stride = stride;
    }
}


void checkResult(int* a, int* b, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (a[i] != b[i])
        {
            printf("Sorting failed!\n");
            return;
        }
    }
    printf("Sorting success!\n");
}


inline long long cpuTimer()
{
    std::chrono::microseconds ms = std::chrono::duration_cast<std::chrono::microseconds>(
        std::chrono::system_clock::now().time_since_epoch()
        );
    return ms.count();
}




/***** Device functions *****/
__global__ void gBitonicSort(int* a, int n_p, bool descending)
{
    unsigned int tid = threadIdx.x;

    int stride_p, half_stride_p, s_p, hs_p, hs, i, j, k, t, hn;
    bool orange;
    hn = 1 << (n_p - 1);
    half_stride_p = 0;
    for (stride_p = 1; stride_p <= n_p; stride_p++)
    {
        s_p = stride_p;
        while (s_p >= 1)
        {
            hs_p = s_p - 1;
            hs = 1 << hs_p;
            for (i = tid; i < hn; i += blockDim.x)
            {
                orange = (i >> half_stride_p) % 2 == 0;
                j = ((i >> hs_p) << s_p) + (i % hs);
                k = j + hs;
                if ((descending && ((orange && a[j] < a[k]) || (!orange && a[j] > a[k]))) ||
                    (!descending && ((orange && a[j] > a[k]) || (!orange && a[j] < a[k]))))
                {
                    t = a[k];
                    a[k] = a[j];
                    a[j] = t;
                }
            }
            __syncthreads();
            s_p = hs_p;
        }
        half_stride_p++;
    }
}




int main(int argc, char** argv)
{
    int p = 14;
    int alg = 0;
    bool descending = false;
    if (argc > 1) p = atoi(argv[1]);
    if (argc > 2) alg = atoi(argv[2]);
    if (argc > 3) descending = (bool)atoi(argv[3]);

    // Number of values
    int n = 1 << p;
    const char* alg_str = alg == 0 ? "hBitonicSortRecursive" : (alg == 1 ? "hBitonicSort" : "gBitonicSort");
    const char* order_str = descending ? "descending" : "ascending";
    printf("Argument 0, number of values:    %d\n", n);
    printf("Argument 1, selected algorithms: %s\n", alg_str);
    printf("Argument 2, descending:          %s\n", order_str);

    // Initialize at host side
    size_t nbytes = n * sizeof(int);
    int* hvals = (int*)malloc(nbytes);
    std::vector<int> hvec(n);
    for (int i = 0; i < n; i++)
    {
        hvals[i] = rand() % n;
        hvec[i] = hvals[i];
    }

    // Sort by STL
    long long t_start = 0, t_elapse = 0;
    t_start = cpuTimer();
    std::sort(hvec.begin(), hvec.end());
    if (descending)
    {
        std::reverse(hvec.begin(), hvec.end());
    }
    t_elapse = cpuTimer() - t_start;
    printf("Time cost on STL sort: %fms\n", (float)t_elapse / 1000);

    // Test algorithms
    switch (alg)
    {
    case 0:
        t_start = cpuTimer();
        hBitonicSortRecursive(hvals, n, descending);
        t_elapse = cpuTimer() - t_start;
        break;
    case 1:
        t_start = cpuTimer();
        hBitonicSort(hvals, n, descending);
        t_elapse = cpuTimer() - t_start;
        break;
    case 2:
    {
        int* dvals = NULL;
        CHECK(hipMalloc((void**)&dvals, nbytes));
        CHECK(hipMemcpy(dvals, hvals, nbytes, hipMemcpyHostToDevice));
        t_start = cpuTimer();
        gBitonicSort << <1, 1024 >> > (dvals, p, descending);
        CHECK(hipDeviceSynchronize());
        t_elapse = cpuTimer() - t_start;
        CHECK(hipMemcpy(hvals, dvals, nbytes, hipMemcpyDeviceToHost));
        CHECK(hipFree(dvals));
        break;
    }
    default:
        printf("Only algorithm 0, 1, 2 were supported!\n");
        break;
    }
    printf("Time cost on %s: %fms\n", alg_str, (float)t_elapse / 1000);
    checkResult(hvals, hvec.data(), n);

    // Free host data
    free(hvals);

    return EXIT_SUCCESS;
}
